/* Copyright (C) 2012  Fabrizio Gueli
 *
 * This file is part of Cuda-complex-sim
 *
 * Cuda-complex-sim is free software: you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation, either
 * version 3 of the License, or (at your option) any later version.
 *
 * Cuda-complex-sim is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with Cuda-complex-sim.  If not, see <http://www.gnu.org/licenses/>.
 */


#include "device.cuh"
#include "host.hpp"
#include "graph_transf.hpp"
#include "h_barabasi_game.hpp"
#include "hygra.cuh"
#include "attributes.hpp"


int main(int argc, char** argv)
{

		bool* nodes_dev;
		Link* links_target_dev;
		uint32_t* mr_array;
		int32_t* counter;
		task_t* task_dev;
		task_arguments* task_args_dev;
		message_t* inbox_dev;
		n_attribute *prog;
		hiprandState *d_state;
		coord* attr_array;


//		if (argc!=3)
	//		{
//				perror("\nErrore");
//				exit(1);
//			}


//	uint32_t max_nodes=atoi(argv[1]);
//	uint8_t average_links=atoi(argv[2]);
	uint16_t supplementary_size= 10;




	uint32_t max_nodes = 1000;
	uint8_t average_links= 6;
	uint16_t barabasi_initial_nodes=2;

	allocateDataStructures(&prog,&nodes_dev, &task_dev, &task_args_dev, &links_target_dev, &inbox_dev,max_nodes,average_links,supplementary_size,&d_state,&mr_array,&counter);
	h_allocateDataStructures(supplementary_size,max_nodes,average_links);

	Graph g = h_barabasi_game(barabasi_initial_nodes, 1, max_nodes);
	generatesCoordinates();
	initAttrArray<coord>(&attr_array);
	copyToDevice(attr_array,(coord*)h_nodes_userattr_array,0,max_nodes);
	startSimulation(links_target_dev,nodes_dev,supplementary_size,g);

	size_t avail;
	size_t total;
	hipMemGetInfo( &avail, &total );
	size_t used = total - avail;
	printf("\nMemoria: totale %d, in uso %d, disponibile: %d", total, used, avail);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	hipEventRecord(start, 0);

	hygra<<<1,64>>>(6);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
	// Clean up:
	hipEventDestroy(start);
	hipEventDestroy(stop);
	FILE *file;
	file=fopen("hygra_times.txt","a");
	fprintf(file, "%f\n",elapsedTime);
	fflush(file);
	fclose(file);

	printf("\nfinito");

	hipDeviceReset();



  return 0;
}
