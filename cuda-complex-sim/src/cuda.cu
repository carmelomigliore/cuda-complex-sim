#include <iostream>

#include "node.hpp"
#include "link.hpp"
#include "parameters.hpp"
#include "message.hpp"

using namespace std;


/*
 * Initializes all data structures on device. Preallocate all needed memory.
 */

__host__ bool allocateDataStructures(Node** nodes_dev_array, Link** links_dev_array, Node*** active_node_dev, Message** message_dev_array, int max_nodes, short max_links, int active_size, short message_buffer){

	if(hipMemcpyToSymbol(HIP_SYMBOL(max_nodes_number), &max_nodes, sizeof(int),0,hipMemcpyHostToDevice)!=hipSuccess){
		cerr << "\nCouldn't allocate memory on device";
		return false;
	}
	if(hipMemcpyToSymbol(HIP_SYMBOL(max_links_number), &max_links, sizeof(int),0,hipMemcpyHostToDevice)!=hipSuccess){
		cerr << "\nCouldn't allocate memory on device";
		return false;
	}
	if(hipMemcpyToSymbol(HIP_SYMBOL(active_nodes_array_size), &active_size, sizeof(int),0,hipMemcpyHostToDevice)!=hipSuccess){
		cerr << "\nCouldn't allocate memory on device";
		return false;
	}
	if(hipMemcpyToSymbol(HIP_SYMBOL(max_links_number), &max_links, sizeof(int),0,hipMemcpyHostToDevice)!=hipSuccess){
		cerr << "\nCouldn't allocate memory on device";
		return false;
	}
	if(hipMalloc((void**)nodes_dev_array,max_nodes*sizeof(Node))!=hipSuccess){
		cerr << "\nCouldn't allocate memory on device";
		return false;
	}
	if(hipMalloc((void**)links_dev_array, max_nodes*max_links*sizeof(Link))!=hipSuccess){
		cerr << "\nCouldn't allocate memory on device";
		return false;
	}
	if(hipMalloc((void***)active_node_dev, active_size*sizeof(Link*))!=hipSuccess){
		cerr << "\nCouldn't allocate memory on device";
		return false;
	}
	if(hipMalloc((void**)message_dev_array, max_nodes*message_buffer*sizeof(Message))!=hipSuccess){
		cerr << "\nCouldn't allocate memory on device";
		return false;
	}
	return true;
}



