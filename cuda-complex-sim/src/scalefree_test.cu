/* Copyright (C) 2012  Fabrizio Gueli
 *
 * This file is part of Cuda-complex-sim
 *
 * Cuda-complex-sim is free software: you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation, either
 * version 3 of the License, or (at your option) any later version.
 *
 * Cuda-complex-sim is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with Cuda-complex-sim.  If not, see <http://www.gnu.org/licenses/>.
 */


#include "device.cuh"
#include "host.hpp"
#include "graph_transf.hpp"
#include "h_barabasi_game.hpp"


int main(int argc, char** argv)
{

		bool* nodes_dev;
		Link* links_target_dev;
		task_t* task_dev;
		task_arguments* task_args_dev;
		message_t* inbox_dev;
		uint32_t* barabasi_links;
		hiprandState *d_state;
		n_attribute *prog;

//		if (argc!=3)
	//		{
//				perror("\nErrore");
//				exit(1);
//			}


//	uint32_t max_nodes=atoi(argv[1]);
//	uint8_t average_links=atoi(argv[2]);
	uint16_t supplementary_size= 10;

	uint16_t barabasi_initial_nodes=3;


	uint32_t max_nodes = 1000000;
	uint8_t average_links= 1;

	allocateDataStructures(&prog,&nodes_dev, &task_dev, &task_args_dev, &links_target_dev, &inbox_dev,  &d_state, &barabasi_links, max_nodes,average_links,supplementary_size,barabasi_initial_nodes);
	h_allocateDataStructures(supplementary_size,max_nodes,average_links);

	Graph g = h_barabasi_game(barabasi_initial_nodes, 1, max_nodes);
	startSimulation(links_target_dev,nodes_dev,supplementary_size,max_nodes,average_links,g);




	srand(time(NULL));
//	Link init;
//	init.target=-1;
//	init_data<<<BLOCKS,THREADS_PER_BLOCK>>>();
//	h_initArray<bool>(false,h_nodes_array,h_max_nodes_number);
//	h_initArray<Link>(init, h_links_target_array, h_max_nodes_number*h_average_links_number);

//	adjlistToCompactList(g);

//	copyToDevice(nodes_dev,h_nodes_array , 0, h_max_nodes_number );
//	copyToDevice(links_target_dev,h_links_target_array ,0, h_max_nodes_number*h_average_links_number );

  	size_t avail;
  	size_t total;
  	hipMemGetInfo( &avail, &total );

  	size_t used = total - avail;
  	printf("\nMemoria: totale %d, in uso %d, disponibile: %d", total, used, avail);

  	hipEvent_t start, stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);
  	// Start record
  	hipEventRecord(start, 0);

  		message_test<<<BLOCKS,THREADS_PER_BLOCK,h_average_links_number*THREADS_PER_BLOCK*sizeof(Link)>>>();
  		message_test2nd<<<BLOCKS,THREADS_PER_BLOCK,h_average_links_number*THREADS_PER_BLOCK*sizeof(Link)>>>();
  		message_test2nd<<<BLOCKS,THREADS_PER_BLOCK,h_average_links_number*THREADS_PER_BLOCK*sizeof(Link)>>>();
  		hipEventRecord(stop, 0);
  		hipEventSynchronize(stop);
  		float elapsedTime;
  		hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
  		// Clean up:
  		hipEventDestroy(start);
  		hipEventDestroy(stop);
  		FILE *file;
  		file=fopen("times.txt","a");
  		fprintf(file, "%f\n",elapsedTime);
  		fflush(file);
  		fclose(file);
  		hipDeviceReset();


  return 0;
}

